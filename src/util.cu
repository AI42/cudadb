#include "util.h"

void setCudaHeapSize(size_t bytes) {
	hipDeviceSetLimit(hipLimitMallocHeapSize, bytes);
}
